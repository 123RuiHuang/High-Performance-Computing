#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include </usr/local/cuda/samples/common/inc/exception.h>
#include </usr/local/cuda/samples/common/inc/helper_string.h>
#include </usr/local/cuda/samples/common/inc/helper_functions.h>
#include </usr/local/cuda/samples/common/inc/hip/hip_runtime_api.h>

unsigned int width,height;
__device__ int partition(unsigned char nums[],int begin, int end) {
	int i = begin - 1;
	unsigned char media = nums[end];
	for(int j = begin; j < end; j++) {
		if(nums[j] < media) {
			i++;
		unsigned	char temp = nums[i];
			nums[i] = nums[j];
			nums[j] = temp;
		}
		
	}
	i++;
	unsigned char temp = nums[i];
	nums[i] = nums[end];
	nums[end] = temp;
  return i;
}

__device__ void quicksort(unsigned char nums[], int begin, int end) {
	if(begin < end) {
		int q = partition(nums,begin,end);
		quicksort(nums,begin,q-1);
		quicksort(nums,q+1,end);
	}

}


__global__ void conv_dev(unsigned char * dev_src, unsigned char * dev_dst, int numRows, int numCols, int KerSize) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    int offset = x + y * gridDim.x * blockDim.x;
    int padding = (KerSize - 1)/2;
    unsigned char *kernel= (unsigned char*)malloc(KerSize*KerSize*sizeof(unsigned char));
   
    if((x >= padding) && (x < (numCols - padding)) && (y >= padding)&& (y < (numRows - padding))) {
        for(int i = 0; i < KerSize; i++) {
          for(int j = 0; j < KerSize; j++) {
             int xx = x + j - padding;
             int yy = y + i - padding;
             int index1 = i*KerSize + j;
             int index2 = xx + yy * gridDim.x * blockDim.x;
             kernel[index1] = dev_src[index2];
            }
           }
           
           // sort the elem in window
  		quicksort(kernel,0,(KerSize * KerSize) - 1);
  		dev_dst[offset] = kernel[padding * KerSize + padding];
  }
		
    free(kernel);
}


void conv(unsigned char * src, unsigned char * dst, int numRows, int numCols, int kerSize) {
    int totalPixels = numRows * numCols;
    unsigned char *dev_src,*dev_dst;
    checkCudaErrors(hipMalloc(&dev_src,sizeof(unsigned char)*totalPixels));
    checkCudaErrors(hipMalloc(&dev_dst,sizeof(unsigned char)*totalPixels));
    dim3 blocks(64,64);
    dim3 threads(8,8);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    checkCudaErrors(hipMemcpy(dev_src,src,sizeof(unsigned char)*totalPixels,hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_dst,src,sizeof(unsigned char)*totalPixels,hipMemcpyHostToDevice));
    
    
    conv_dev<<<blocks,threads>>>(dev_src,dev_dst,numRows,numCols,kerSize);
    checkCudaErrors(hipMemcpy(dst,dev_dst,sizeof(unsigned char)*totalPixels,hipMemcpyDeviceToHost));
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("filter size: %d, time to copy-compute-copy : %f second\n", kerSize,elapsedTime / 1000.0);
    printf("\n");
    checkCudaErrors(hipFree(dev_src));
    checkCudaErrors(hipFree(dev_dst));
    
}







int main(int argc,char* argv[]) {
 int winSize = atoi(argv[1]);
 const char *source_image_Path = argv[2];
 const char *dst_image_Path =  argv[3];
 unsigned char* image_source = NULL;
 unsigned char* image_dst = NULL;  // store the result calculated at GPU
 sdkLoadPGM(source_image_Path,&image_source,&width,&height);
 image_dst = (unsigned char*)malloc(width*height*sizeof(unsigned char));
 conv(image_source,image_dst,width,height,winSize);
 sdkSavePGM(dst_image_Path,image_dst,width,height);
 free(image_dst);
 printf("\n");
}

