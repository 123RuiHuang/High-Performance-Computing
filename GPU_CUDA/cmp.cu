#include <stdio.h>
#include <string.h>
#include </usr/local/cuda/samples/common/inc/exception.h>
#include </usr/local/cuda/samples/common/inc/helper_string.h>
#include </usr/local/cuda/samples/common/inc/helper_functions.h>
#include </usr/local/cuda/samples/common/inc/hip/hip_runtime_api.h>
unsigned int width,height;
int main() {
  const char * img_GPU_Path = NULL;
  const char * img_CPU_Path = NULL;

 for(int i = 0; i < 4; i++) {
   if(i == 0) {
    img_GPU_Path = "lena_GPU_3.pgm";
    img_CPU_Path = "lena_CPU_3.pgm";
   }
   if(i == 1) {
     img_GPU_Path = "lena_GPU_7.pgm";
     img_CPU_Path = "lena_CPU_7.pgm";
   }
   if(i == 2) {
     img_GPU_Path = "lena_GPU_11.pgm";
     img_CPU_Path = "lena_CPU_11.pgm";
   }
   if(i == 3) {
     img_GPU_Path = "lena_GPU_15.pgm";
     img_CPU_Path = "lena_CPU_15.pgm";
     }
     
   int counter1 = 0;
   int counter2 = 0;
   unsigned char* img_GPU = NULL;
   unsigned char* img_CPU = NULL;  
   sdkLoadPGM(img_GPU_Path,&img_GPU,&width,&height);
   sdkLoadPGM(img_CPU_Path,&img_CPU,&width,&height);
   for(int m = 0; m < height; m++) {
     for(int n = 0; n < width; n++) {
       if(img_GPU[m * width + n] != img_CPU[m * width + n]) counter1++;
       if(img_GPU[m * width + n] == img_CPU[m * width + n]) counter2++;
     }
   }
   printf("filter size : %d ; Accuracy : %f\n",4*(i+1) - 1,(float)counter2 / (counter1+counter2));
 }
 
}