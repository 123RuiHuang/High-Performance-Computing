#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <time.h>
#include </usr/local/cuda/samples/common/inc/helper_string.h>
#include </usr/local/cuda/samples/common/inc/helper_functions.h>
#include </usr/local/cuda/samples/common/inc/hip/hip_runtime_api.h>


unsigned int width,height;

int partition(unsigned char nums[],int begin, int end) {
	int i = begin - 1;
	unsigned char media = nums[end];
	for(int j = begin; j < end; j++) {
		if(nums[j] < media) {
			i++;
		unsigned	char temp = nums[i];
			nums[i] = nums[j];
			nums[j] = temp;
		}
		
	}
	i++;
	unsigned char temp = nums[i];
	nums[i] = nums[end];
	nums[end] = temp;
  return i;
}


void quicksort(unsigned char nums[], int begin, int end) {
	if(begin < end) {
		int q = partition(nums,begin,end);
		quicksort(nums,begin,q-1);
		quicksort(nums,q+1,end);
	}

}






void conv(unsigned char * src, unsigned char * dst, int numRows, int numCols, int kerSize) {
        
        for(int i = 0; i < numRows;i++) {// copy src to dst
          for(int j = 0; j < numCols; j++) {
              dst[i * numCols + j] = src[i * numCols + j];
          }
        }
        unsigned char *kernel= (unsigned char*)malloc(kerSize*kerSize*sizeof(unsigned char));
        int padding = (kerSize - 1)/2;
        for(int i = 0; i < numRows; i++) { //ith row
          for(int j = 0; j < numCols;j++) { //jth column
            if((i >= padding) && (i < (numRows - padding)) && (j >= padding)&& (j < (numCols - padding))) {
                  for(int ii = 0; ii < 2 * padding + 1; ii++) {
                    for(int jj = 0; jj <  2 * padding + 1; jj++) {
                        kernel[ii * kerSize + jj] = src[(i-padding + ii)*numCols + (j-padding +jj)];
                       // printf("%d \n",ii * kerSize + jj);
                    }
                  }
                  quicksort(kernel,0,(kerSize * kerSize) - 1);
                  dst[i*numCols + j] = kernel[padding * kerSize + padding];
            }
          }
        }
        free(kernel);
}





int main(int argc,char* argv[]) {
 int winSize = atoi(argv[1]);
 const char *source_image_Path = argv[2];
 const char *dst_image_Path =  argv[3];
 unsigned char* image_source = NULL;
 unsigned char* image_dst = NULL;  // store the result calculated at CPU
 sdkLoadPGM(source_image_Path,&image_source,&width,&height);

 image_dst = (unsigned char*)malloc(width*height*sizeof(unsigned char));
 clock_t start = clock();
 conv(image_source,image_dst,width,height,winSize);
 clock_t end = clock();
 printf("filter size : %d, time : %f second\n", winSize, (double)(end-start)/CLOCKS_PER_SEC);
 sdkSavePGM(dst_image_Path,image_dst,width,height);
 free(image_dst);
 printf("\n");

}

